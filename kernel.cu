#include "hip/hip_runtime.h"
﻿#include ""
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// include hiprand
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <stdlib.h>
#include <ctype.h>
#include <stdio.h>
#include <vector>
#define SHA256_BLOCK_SIZE 32 // SHA256 outputs a 32 byte digest

#define ROTLEFT(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
#define ROTRIGHT(a, b) (((a) >> (b)) | ((a) << (32 - (b))))
#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ ((x) >> 10))
#define BCD(c) 5 * (5 * (5 * (5 * (5 * (5 * (5 * (5*(5*(c&512)+(c&256))+(c&128))+(c&64))+(c&32))+(c&16))+(c&8))+(c&4))+(c&2))+(c&1)
/**************************** DATA TYPES ****************************/
typedef unsigned char BYTE; // 8-bit byte
typedef uint32_t WORD;		// 32-bit word, change to "long" for 16-bit machines

typedef struct JOB
{
	BYTE* data;
	unsigned long long size;
	BYTE digest[64];
} JOB;

typedef struct
{
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[8];
} SHA256_CTX;

__constant__ WORD dev_k[64];

static const WORD host_k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

/*********************** FUNCTION DECLARATIONS **********************/
char* print_sha(BYTE* buff);
__device__ void sha256_init(SHA256_CTX* ctx);
__device__ void sha256_update(SHA256_CTX* ctx, const BYTE data[], size_t len);
__device__ void sha256_final(SHA256_CTX* ctx, BYTE hash[]);
__device__ int isspace(unsigned char c) {
	return c == ' ' || c == '\t' || c == '\n' || c == '\r' || c == '\f' || c == '\v';
}
__device__ char* trim(char* str) {
	size_t len = 0;
	char* frontp = str;
	char* endp = NULL;

	if (str == NULL) { return NULL; }
	if (str[0] == '\0') { return str; }
	for (int len = 0; str[len] != '\0'; len++) {
		if (str[len] != ' ') {
			endp = str + len;
			break;
		}
	}
	endp = str + len;

	/* Move the front and back pointers to address the first non-whitespace
	 * characters from each end.
	 */
	while (isspace((unsigned char)*frontp)) { ++frontp; }
	if (endp != frontp)
	{
		while (isspace((unsigned char)*(--endp)) && endp != frontp) {}
	}

	if (str + len - 1 != endp)
		*(endp + 1) = '\0';
	else if (frontp != str && endp == frontp)
		*str = '\0';

	/* Shift the string so that it starts at str so that if it's dynamically
	 * allocated, we can still free it on the returned pointer.  Note the reuse
	 * of endp to mean the front of the string buffer now.
	 */
	endp = str;
	if (frontp != str)
	{
		while (*frontp) { *endp++ = *frontp++; }
		*endp = '\0';
	}


	return str;
}


char* hash_to_string(BYTE* buff)
{
	char* string = (char*)malloc(70);
	int k, i;
	for (i = 0, k = 0; i < 32; i++, k += 2)
	{
		sprintf(string + k, "%.2x", buff[i]);
		// printf("%02x", buff[i]);
	}
	string[64] = 0;
	return string;
}

__device__ void sha256_transform(SHA256_CTX* ctx, const BYTE data[])
{

	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];
	WORD S[8];

	//mycpy32(S, ctx->state);

#pragma unroll 16
	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);

#pragma unroll 64
	for (; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];

#pragma unroll 64
	for (i = 0; i < 64; ++i) {
		t1 = h + EP1(e) + CH(e, f, g) + dev_k[i] + m[i];
		t2 = EP0(a) + MAJ(a, b, c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;

}

__device__ void sha256_init(SHA256_CTX* ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void sha256_update(SHA256_CTX* ctx, const BYTE data[], size_t len)
{
	WORD i;

	// for each byte in message
	for (i = 0; i < len; ++i) {
		// ctx->data == message 512 bit chunk
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void sha256_final(SHA256_CTX* ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}

}

#define checkCudaErrors(x)                                                    \
	{                                                                         \
		hipGetLastError();                                                   \
		x;                                                                    \
		hipError_t err = hipGetLastError();                                 \
		if (err != hipSuccess)                                               \
			printf("GPU: hipError_t %d (%s)\n", err, hipGetErrorString(err)); \
	}
__global__ void sha256_cuda(BYTE* data, int len, BYTE* result)
{
	//source: src_block
	// 　　　　不定長
	//result: nans&hash(sha256)
	//      2進化10進数整数(10byte)　32byte
	//　　　　↑固定長↑
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//文字列に乱数を挿入する。
	BYTE* mae = (BYTE*)malloc((len + 10) * sizeof(BYTE));
	mae = (BYTE*)malloc((len + 10) * sizeof(BYTE));
	int conma = 0;
	int index_ = 0;
	//,が２回出現したらその場所をindex_に保存してループを抜ける。
	for (index_ = 0; /*index_ < len*/; index_++)
	{
		mae[index_] = (BYTE)data[index_];
		if (data[index_] == ',')
		{
			conma += 1;
			if (conma == 2)
				break;
		}
	}
	hiprandState s;

	hiprand_init((unsigned long long)clock64() + i, 0, 0, &s);

	BYTE* str = (BYTE*)malloc(32);
	int rand = hiprand_uniform(&s) * 1000000000;
	//数字を文字列にしちゃうやつ。
	int da = 0;
	for (da = 0; rand != 0; da++)
	{
		int rem = rand % 10;
		str[da] = (BYTE)(rem + '0');
		rand = rand / 10;
	}

	int strlen = 0;
	for (strlen = 0; str[strlen] != '\0'; strlen++);
	//乱数の文字数を取得する。
	for (int a = 0; a < strlen; a++)
	{ // maeにstrをくっつける
		mae[index_ + a + 1] = str[a];
	}
	// maeに残りのdata[i]をくっつける。
	for (;/* index_ < len*/; index_++)
	{
		if (data[index_] != '\0') {
			mae[index_ + strlen + 1] = data[index_ + 1];
		}
		else {
			break;
		}
	}
	mae[index_ + strlen] = '\0';
	printf("%s←\n", mae);

	// help!!!!!
	SHA256_CTX ctx;
	sha256_init(&ctx);
	printf("%i\n", index_ + strlen);
	sha256_update(&ctx, mae, index_ + strlen);
	BYTE digest[64];
	for (int a = 0; a < 64; a++)
	{
		digest[a] = 0xff;
	}
	sha256_final(&ctx, (digest));
	//result: ,nans&hash(sha256),nans&hash(sha256)
	//TODO
	printf("%c\r\n",digest[0]);
	result[i * (16 + 2 + 9)] = ',';
	for (int a = i * (16 + 2 + 9) + 1, int b = 0; b < strlen; b++, a++) {
		result[a] = str[b];
	}
	result[i * (16 + 2 + 9)+strlen+1] = '&';
	for (int a = i * (16 + 4 + 9) + strlen + 2, int b = 0; b < 32; b++,a++) {
		result[a] = digest[b];
	}
}
int main(int argc, char** argv)
{
	//source: src_block
	// 　　　　不定長
	//result: hash(sha256)&nans
	//      2進化10進数整数(10byte)　32byte
	//　　　　↑固定長↑
	int threads = 10;
	hipSetDevice(0);
	BYTE* data1 = reinterpret_cast<BYTE*>("00000193ab920406e5586e1a99472d573138e27191ca77ca897dc400f9abc8b8,b0bb15df4e3b489c5601b6a9c2d1aea66396b992653f547c22662a69e21bc8ec,,101,1657528305862");
	BYTE* dev_data;
	BYTE* dev_result_str;
	hipMallocManaged((void**)&dev_data, 600 * sizeof(BYTE));
	hipMallocManaged((void**)&dev_result_str, (32 + 2 + 9) * threads * sizeof(BYTE));
	hipMemcpy(dev_data, data1, sizeof(BYTE) * 600, hipMemcpyHostToDevice);
	sha256_cuda << <1, threads >> > (dev_data, 600, dev_result_str);
	hipDeviceSynchronize();
	BYTE* result_str;
	result_str = (BYTE*)malloc((16 + 2 + 9) * threads * sizeof(BYTE));
	hipMemcpy(result_str, dev_result_str, (16 + 2 + 9) * threads * sizeof(BYTE), hipMemcpyDeviceToHost);
	for(int a = 0; a < (16 + 2 + 9) * threads; a++)
	{
		printf("%c", result_str[a]);
	}
	return 0;
}