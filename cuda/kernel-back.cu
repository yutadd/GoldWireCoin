#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// include hiprand
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <stdlib.h>
#include <ctype.h>
#include <stdio.h>
#include <vector>
#define SHA256_BLOCK_SIZE 32 // SHA256 outputs a 32 byte digest

#define ROTLEFT(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
#define ROTRIGHT(a, b) (((a) >> (b)) | ((a) << (32 - (b))))

#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ ((x) >> 10))

/**************************** DATA TYPES ****************************/
typedef unsigned char BYTE; // 8-bit byte
typedef uint32_t WORD;		// 32-bit word, change to "long" for 16-bit machines

typedef struct JOB
{
	BYTE* data;
	unsigned long long size;
	BYTE digest[64];
} JOB;

typedef struct
{
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[8];
} SHA256_CTX;

__constant__ WORD dev_k[64];

static const WORD host_k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

/*********************** FUNCTION DECLARATIONS **********************/
char* print_sha(BYTE* buff);
__device__ void sha256_init(SHA256_CTX* ctx);
__device__ void sha256_update(SHA256_CTX* ctx, const BYTE data[], size_t len);
__device__ void sha256_final(SHA256_CTX* ctx, BYTE hash[]);

char* hash_to_string(BYTE* buff)
{
	char* string = (char*)malloc(70);
	int k, i;
	for (i = 0, k = 0; i < 32; i++, k += 2)
	{
		sprintf(string + k, "%.2x", buff[i]);
		// printf("%02x", buff[i]);
	}
	string[64] = 0;
	return string;
}

__device__ void sha256_transform(SHA256_CTX* ctx, const BYTE data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];
	WORD S[8];

	// mycpy32(S, ctx->state);

#pragma unroll 16
	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);

#pragma unroll 64
	for (; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];

#pragma unroll 64
	for (i = 0; i < 64; ++i)
	{
		t1 = h + EP1(e) + CH(e, f, g) + dev_k[i] + m[i];
		t2 = EP0(a) + MAJ(a, b, c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

__device__ void sha256_init(SHA256_CTX* ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void sha256_update(SHA256_CTX* ctx, const BYTE data[], size_t len)
{
	WORD i;
	for (i = 0; i < len; ++i)
	{
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64)
		{
			sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void sha256_final(SHA256_CTX* ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;
	if (ctx->datalen < 56)
	{
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else
	{
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha256_transform(ctx, ctx->data);

	for (i = 0; i < 4; ++i)
	{
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}

#define checkCudaErrors(x)                                                    \
	{                                                                         \
		hipGetLastError();                                                   \
		x;                                                                    \
		hipError_t err = hipGetLastError();                                 \
		if (err != hipSuccess)                                               \
			printf("GPU: hipError_t %d (%s)\n", err, hipGetErrorString(err)); \
	}
__global__ void sha256_cuda(char* data, int len, long long int* result_nans, BYTE* result)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//文字列に乱数を挿入する。
	char* mae = "";
	int conma = 0;
	int index_ = 0;
	//,が２回出現したらその場所をindex_に保存してループを抜ける。
	for (index_ = 0; index_ < len; index_++)
	{
		mae[index_] = (char)data[index_];
		if (data[index_] == ',')
		{
			conma += 1;
			if (conma == 2)
				break;
		}
	}
	hiprandState s;

	hiprand_init((unsigned long long)clock64() + i, 0, 0, &s);

	char* str = (char*)malloc(32);
	int rand = hiprand_uniform(&s) * 1000000000;
	result_nans[i] = rand;
	//数字を文字列にしちゃうやつ。
	int da = 0;
	for (da = 0; rand != 0; da++)
	{
		int rem = rand % 10;
		str[da] = (char)(rem + '0');
		rand = rand / 10;
	}
	int srclen = 0;
	for (srclen = 0; str[srclen] != '\0'; srclen++);
	//乱数の文字数を取得する。
	for (int a = 0; a < srclen; a++)
	{ // maeにsrcをくっつける
		mae[index_ + a] = str[a];
	}
	// maeに残りのdata[i]をくっつける。
	for (index_ += srclen; index_ < sizeof(data[i]) / sizeof(char); index_++)
	{
		mae[index_ + srclen] = data[index_];
	}
	// help!!!!!

	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, (unsigned char*)mae, len + srclen);
	BYTE* digest = (BYTE*)malloc(64 * sizeof(BYTE));
	for (int a = 0; a < 64; a++)
	{
		digest[a] = 0xff;
	}

	sha256_final(&ctx, (digest));
	char* string = (char*)malloc(70);
	int k;
	for (int a = 0, k = 0; a < 32; a++, k += 2)
	{
		string[k] = (char)digest[a];
	}

	
	for (int a = 0; a<70;a++) {
		result[a] = string[a];
	}
}
int main(int argc, char** argv)
{
	hipSetDevice(0);
	char* data1 = "previous_hash,addr,,1";
	char* dev_data;
	BYTE* dev_result_str;
	long long int* dev_result;
	hipMallocManaged((void**)&dev_data, 100 * sizeof(char));
	hipMallocManaged((void**)&dev_result, sizeof(long long int));
	hipMallocManaged((void**)&dev_result_str, sizeof(char*) * 70);
	hipMemcpy(dev_data, data1, 100 * sizeof(char), hipMemcpyHostToDevice);
	char* buff = (char*)malloc(sizeof(long long int));
	sha256_cuda << <1, 1 >> > (dev_data, 22, dev_result, dev_result_str);
	hipDeviceSynchronize();
	long long int *result;

	result = (long long int *)malloc(1 * sizeof(long long int));
	hipMemcpy(result, dev_result, sizeof(long long int), hipMemcpyDeviceToHost);
	printf("%lld\n", result[0]);
	BYTE* result_str;
	result_str = (BYTE*)malloc(70);
	hipMemcpy(result_str, dev_result_str, 70, hipMemcpyDeviceToHost);
	printf( hash_to_string(result_str));
	return 0;
}